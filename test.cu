#include "hip/hip_runtime.h"
#include <cstdio>
__global__ void add(int * dev_a[], int * dev_b[], int * dev_c[])
{
  int i;
  int j;
  j = blockIdx.x * blockDim.x + threadIdx.x;
  i = blockIdx.y * blockDim.y + threadIdx.y;
  dev_c[i][j]=dev_a[i][j]+dev_b[i][j];
}

inline void GPUassert(hipError_t code, char * file, int line, bool Abort=true)
{
    if (code != 0) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code),file,line);
        if (Abort) exit(code);
    }       
}

#define GPUerrchk(ans) { GPUassert((ans), __FILE__, __LINE__); }

int main(void)
{
    const int aa[2][2]={{1,2},{3,4}};
    const int bb[2][2]={{5,6},{7,8}};
    int cc[2][2];

    int ** h_a = (int **)malloc(2 * sizeof(int *));
    for(int i=0; i<2;i++){
        GPUerrchk(hipMalloc((void**)&h_a[i], 2*sizeof(int)));
        GPUerrchk(hipMemcpy(h_a[i], &aa[i][0], 2*sizeof(int), hipMemcpyHostToDevice));
    }

    int **d_a;
    GPUerrchk(hipMalloc((void ***)&d_a, 2 * sizeof(int *)));
    GPUerrchk(hipMemcpy(d_a, h_a, 2*sizeof(int *), hipMemcpyHostToDevice));

    int ** h_b = (int **)malloc(2 * sizeof(int *));
    for(int i=0; i<2;i++){
        GPUerrchk(hipMalloc((void**)&h_b[i], 2*sizeof(int)));
        GPUerrchk(hipMemcpy(h_b[i], &bb[i][0], 2*sizeof(int), hipMemcpyHostToDevice));
    }

    int ** d_b;
    GPUerrchk(hipMalloc((void ***)&d_b, 2 * sizeof(int *)));
    GPUerrchk(hipMemcpy(d_b, h_b, 2*sizeof(int *), hipMemcpyHostToDevice));

    int ** h_c = (int **)malloc(2 * sizeof(int *));
    for(int i=0; i<2;i++){
        GPUerrchk(hipMalloc((void**)&h_c[i], 2*sizeof(int)));
    }

    int ** d_c;
    GPUerrchk(hipMalloc((void ***)&d_c, 2 * sizeof(int *)));
    GPUerrchk(hipMemcpy(d_c, h_c, 2*sizeof(int *), hipMemcpyHostToDevice));
    dim3 threads(2,2);
    dim3 blocks(1,1);

    add<<<threads,blocks>>>(d_a,d_b,d_c);
    GPUerrchk(hipPeekAtLastError());

    for(int i=0; i<2;i++){
        GPUerrchk(hipMemcpy(&cc[i][0], h_c[i], 2*sizeof(int), hipMemcpyDeviceToHost));
    }

    for(int i=0;i<2;i++) {
        for(int j=0;j<2;j++) {
            printf("(%d,%d):%d\n",i,j,cc[i][j]);
        }
    }

    return hipDeviceReset();
}
